#include <stdio.h>
#include <stdlib.h>
#include "common.h"
#include "params.h"
#include "io.h"
#include <mpi.h>
#include <pthread.h>

/* for timing */
#include <sys/time.h>
inline double seconds()
{
    struct timeval tp;
    struct timezone tzp;
    int i = gettimeofday(&tp, &tzp);
    return ((double)tp.tv_sec + (double)tp.tv_usec * 1.e-6);
}
/* for timing */

extern void get_params(int argc, char **argv);
extern void print_params();
extern void set_device_params();

//extern void alloc_metric(double *M);
extern void alloc_wave(Wave *);
extern void alloc_wave_halo();
extern void alloc_fault(Fault *);                  
extern void alloc_fault_coef(Fault *);             
extern void alloc_pml(PML *);
extern void alloc_pml_host(PML *);
extern void dealloc_wave(Wave);
extern void dealloc_fault(Fault);
extern void dealloc_fault_coef(Fault);
extern void dealloc_pml_host(PML);
extern void dealloc_pml(PML);
extern void cpy_host2device_pml(PML, const PML);

extern void get_coord_y_h(realptr_t C, realptr_t hC, const int j);
extern void get_coord_x_h(realptr_t C, realptr_t hC, const int i);
extern void construct_coord(realptr_t C);
extern void cal_metric(realptr_t C, realptr_t M);
extern void extend_Symm_array(realptr_t W, int SIZE);
extern void extend_crew_array(realptr_t W, int SIZE);
extern void abs_init(PML);
extern void init_cerjan_host(realptr_t damp);

extern void init_media1d(realptr_t C, realptr_t M);
extern void init_media3d(realptr_t C, realptr_t M);

extern void exchange_array(realptr_t, int arrsize);
extern void exchange_wave(realptr_t);
extern void exchange_fault(Fault, int);               //*******

extern void init_wave_3d(realptr_t);
extern void init_fault(Fault, int, int Faultgrid[]);                   //*******
extern void trial_sw(Wave, Fault, realptr_t M, int it, int irk, int, int, int, int, int, int Faultgrid[]);       //
extern void trial_rs(Wave, Fault, realptr_t M, int it, int irk, int, int, int, int, int, int Faultgrid[]);       //
extern void thermpress(Wave, Fault, realptr_t M, int it, int irk, int, int, int, int);          //
extern void wave2fault(Wave, Fault, realptr_t M, int, int);         //    
extern void fault2wave(Wave, Fault, realptr_t M, int, int);         //
extern void wave_deriv (Wave, realptr_t M, PML, int, int, int);
extern void abs_deriv_x(Wave, realptr_t M, PML, int, int);
extern void abs_deriv_y(Wave, realptr_t M, PML, int, int);
extern void abs_deriv_z(Wave, realptr_t M, PML, int, int);
//extern void fault_deriv(Wave, Fault, realptr_t M, int, int, int);      
extern void fault_dvelo(Wave, Fault, realptr_t M, int, int, int, int, int);          //
extern void fault_dstrs_f(Wave, Fault, realptr_t M, int, int, int, int, int);         //
extern void fault_dstrs_b(Wave, Fault, realptr_t M, int, int, int, int, int);         //
extern void wave_rk(Wave, PML, int irk);
extern void fault_rk(Fault, int irk, int);                         //
extern void state_rk(Fault, int irk, int, int Faultgrid[]);                         //
extern void init_fault_coef(realptr_t M, Fault, int, int);            //************
extern void init_wave_free(realptr_t M, PML, Wave);
extern void smooth_gauss_volume(Wave, Fault, realptr_t M, int, int, int Faultgrid[], int);   //
extern void smooth_T1(Fault, int, int Faultgrid[]);
//extern void fault_filter(Wave, Fault, realptr_t M);
extern void nc_read_init_stress(Fault F);                         

extern void cal_range_steph(realptr_t C, real_t *range);
extern void cal_range_media(realptr_t M, real_t *range);

extern void add_source_ricker(Wave W, realptr_t M, int it, int irk);

extern void apply_cerjan(Wave, realptr_t);

extern void cal_rup_sensor(Fault, int, int Faultgrid[]);                //*********

int FD_Flags[8][3] = {
{-1, -1, -1},
{ 1,  1, -1},
{ 1,  1,  1},
{-1, -1,  1},
{-1,  1, -1},
{ 1, -1, -1},
{ 1, -1,  1},
{-1,  1,  1}}; // -1(B) 1(F)
//int FD_Flags[8][3] = {
//{ 1,  1,  1},
//{ 1,  1,  1},
//{ 1,  1,  1},
//{ 1,  1,  1},
//{ 1,  1,  1},
//{ 1,  1,  1},
//{ 1,  1,  1},
//{ 1,  1,  1}}; // -1(B) 1(F)

void setDeviceBeforeInit(){
  char *localRankStr = NULL;
  int rank = 0, devCount = 0;
  if (NULL != (localRankStr = getenv(ENV_LOCAL_RANK))){
    rank = atoi(localRankStr);
  }
  CUDACHECK(hipGetDeviceCount( &devCount));
  if(masternode) printf("There are %d GPUs on each node\n", devCount);
  CUDACHECK(hipSetDevice( (rank + 0) % devCount));
}

void Initialize(int *argc, char ***argv, int *rank, int *size){
  setDeviceBeforeInit();
  MPI_Init(argc, argv);
  MPI_Comm_rank(MPI_COMM_WORLD, rank);
  MPI_Comm_size(MPI_COMM_WORLD, size);
}

int main(int argc, char **argv){

  int rank, size, dims[3];
  int reorder = 1; // true
  int periods[3] = {0, 0, 0}; // false
  int ilen = 0;
  int* Faultgrid;
  
  char processor[MPI_MAX_PROCESSOR_NAME];
  
  Initialize(&argc, &argv, &rank, &size);  
  MPI_Get_processor_name(processor, &ilen);

  get_params(argc, argv);

  if(hostParams.PX*hostParams.PY*hostParams.PZ < 2){
    hipSetDevice(hostParams.igpu);
  }

  int oldRank = rank;

  dims[0] = hostParams.PX;
  dims[1] = hostParams.PY;
  dims[2] = hostParams.PZ;
  
  MPI_Cart_create(MPI_COMM_WORLD, 3, dims, periods, reorder, &SWMPI_COMM);
  MPI_Comm_rank(SWMPI_COMM, &rank);

  if(rank != oldRank){
    printf("Rank change: from %d to %d\n", oldRank, rank);
  }

  MPI_Cart_coords(SWMPI_COMM, rank, 3, thisid);
  MPI_Cart_shift(SWMPI_COMM, 0, 1, &neigxid[0], &neigxid[1]);
  MPI_Cart_shift(SWMPI_COMM, 1, 1, &neigyid[0], &neigyid[1]);
  MPI_Cart_shift(SWMPI_COMM, 2, 1, &neigzid[0], &neigzid[1]);

  if(0 == rank) masternode = 1;
  if(masternode) print_params();

  int absnode[6] = {0, 0, 0, 0, 0, 0};

  if(neigxid[0] == MPI_PROC_NULL) absnode[0] = 1;
  if(neigxid[1] == MPI_PROC_NULL) absnode[1] = 1;
  if(neigyid[0] == MPI_PROC_NULL) absnode[2] = 1;
  if(neigyid[1] == MPI_PROC_NULL) absnode[3] = 1;
  if(neigzid[0] == MPI_PROC_NULL) absnode[4] = 1;
  if(neigzid[1] == MPI_PROC_NULL){
#ifdef FreeSurface
    absnode[5] = 0; freenode = 1;
#else
    absnode[5] = 1; freenode = 0;
#endif
  }

  hostParams.freenode = freenode;
  // hostParams.faultnode = faultnode;

  hostParams.rankx = thisid[0];
  hostParams.ranky = thisid[1];
  hostParams.rankz = thisid[2];

  set_device_params(); // copy parameters from host to device
  hipDeviceSynchronize();

  int ni = hostParams.ni;
  int nj = hostParams.nj;
  int nk = hostParams.nk;

  int nx = hostParams.nx;
  int ny = hostParams.ny;
  int nz = hostParams.nz;

  int i0 = 0;
  int num_fault = hostParams.num_fault;
  int faultnode = 0;
  // int* faultnode_Dev;
  //int srci = hostParams.NX / 2; // fault plane index X
  for(int nfault = 0; nfault < num_fault; nfault++){
    i0 = hostParams.src_i[nfault];
    if(i0 / ni == thisid[0]) faultnode = 1;
    // hostParams.faultnode = faultnode;
  }

  // hipMalloc(&faultnode_Dev, sizeof(int)*1);
  // hipMemcpy(faultnode_Dev, faultnode, sizeof(int)*1, hipMemcpyHostToDevice);

  MPI_Barrier(MPI_COMM_WORLD);

  realptr_t M;  // device
  Wave  W;   // device
  Fault F;   // device
  PML   P;   // device
  PML   h_P; // host
  Recv R; // host


  h_P.isx1 = absnode[0];
  h_P.isx2 = absnode[1];
  h_P.isy1 = absnode[2];
  h_P.isy2 = absnode[3];
  h_P.isz1 = absnode[4];
  h_P.isz2 = absnode[5];

  P.isx1 = absnode[0];
  P.isx2 = absnode[1];
  P.isy1 = absnode[2];
  P.isy2 = absnode[3];
  P.isz1 = absnode[4];
  P.isz2 = absnode[5];

  MPI_Barrier(MPI_COMM_WORLD);
  printf("(%s): %d %d %d masternode = %d, freenode = %d\n", processor,
      thisid[0],
      thisid[1],
      thisid[2],
      masternode, freenode);
 // MPI_Barrier(MPI_COMM_WORLD);
  printf("(%s): %d %d %d absnode = (%d %d) (%d %d) (%d %d)\n", processor,
      thisid[0],
      thisid[1],
      thisid[2],
      absnode[0],
      absnode[1],
      absnode[2],
      absnode[3],
      absnode[4],
      absnode[5]);
  MPI_Barrier(MPI_COMM_WORLD);

  hipMalloc((void**) &M, sizeof(real_t)*nx*ny*nz*MSIZE);
  alloc_wave(&W);
  alloc_fault(&F);
  alloc_fault_coef(&F);

  alloc_wave_halo();

#ifdef usePML
  alloc_pml_host(&h_P);
  abs_init(h_P);
  //coef_surface(P, M);
  alloc_pml(&P);
  cpy_host2device_pml(P, h_P);
#endif

#ifdef useCerjan
  if (masternode) printf("Using Cerjan (1985) for absorbing ...\n");

  real_t *damp_h = (real_t *) malloc(sizeof(real_t)*nx*ny*nz);
  memset(damp_h, 0, sizeof(real_t)*nx*ny*nz);
  real_t *damp;
  hipMalloc((real_t **) &damp, sizeof(real_t)*nx*ny*nz);

  init_cerjan_host(damp_h);
  hipMemcpy(damp, damp_h, sizeof(real_t)*nx*ny*nz, hipMemcpyHostToDevice);
#endif

  CUDACHECK(hipDeviceSynchronize());
  CUDACHECK(hipGetLastError());

  realptr_t hostCoord, hostMetric;
  hostCoord  = (real_t *)malloc(sizeof(real_t)*nx*ny*nz*CSIZE);
  hostMetric = (real_t *)malloc(sizeof(real_t)*nx*ny*nz*MSIZE);
  memset(hostCoord,  0, sizeof(real_t)*nx*ny*nz*CSIZE);
  memset(hostMetric, 0, sizeof(real_t)*nx*ny*nz*MSIZE);

  construct_coord(hostCoord);
  extend_crew_array(hostCoord, CSIZE);
  MPI_Barrier(MPI_COMM_WORLD);
  exchange_array(hostCoord, CSIZE);
  MPI_Barrier(MPI_COMM_WORLD);

  //locate_recv(&R,hostCoord);

  real_t range[2];
  cal_range_steph(hostCoord, range);
  real_t hmin = range[0];
  real_t hmax = range[1];
  real_t hmin_global, hmax_global;
  MPI_Barrier(MPI_COMM_WORLD);
  MPI_Reduce(&hmin, &hmin_global, 1, MPI_REAL_T, MPI_MIN, 0, MPI_COMM_WORLD);
  MPI_Reduce(&hmax, &hmax_global, 1, MPI_REAL_T, MPI_MAX, 0, MPI_COMM_WORLD);

  cal_metric(hostCoord, hostMetric);
  if (flag_media1d) init_media1d(hostCoord, hostMetric);
  if (flag_media3d) init_media3d(hostCoord, hostMetric);

  extend_Symm_array(hostMetric, MSIZE);
  MPI_Barrier(MPI_COMM_WORLD);
  exchange_array(hostMetric, MSIZE);
  hipMemcpy(M, hostMetric, sizeof(real_t)*nx*ny*nz*MSIZE, hipMemcpyHostToDevice);
  //free(hostCoord);
  //free(hostMetric);

  real_t media_range[6];
  cal_range_media(hostMetric, media_range);
  real_t  vp_min = media_range[0];
  real_t  vp_max = media_range[1];
  real_t  vs_min = media_range[2];
  real_t  vs_max = media_range[3];
  real_t rho_min = media_range[4];
  real_t rho_max = media_range[5];
  real_t vp_min_global, vs_min_global, rho_min_global;
  real_t vp_max_global, vs_max_global, rho_max_global;
  MPI_Barrier(MPI_COMM_WORLD);
  MPI_Reduce(&vp_min, &vp_min_global, 1, MPI_REAL_T, MPI_MIN, 0, MPI_COMM_WORLD);
  MPI_Reduce(&vp_max, &vp_max_global, 1, MPI_REAL_T, MPI_MAX, 0, MPI_COMM_WORLD);
  MPI_Reduce(&vs_min, &vs_min_global, 1, MPI_REAL_T, MPI_MIN, 0, MPI_COMM_WORLD);
  MPI_Reduce(&vs_max, &vs_max_global, 1, MPI_REAL_T, MPI_MAX, 0, MPI_COMM_WORLD);
  MPI_Reduce(&rho_min, &rho_min_global, 1, MPI_REAL_T, MPI_MIN, 0, MPI_COMM_WORLD);
  MPI_Reduce(&rho_max, &rho_max_global, 1, MPI_REAL_T, MPI_MAX, 0, MPI_COMM_WORLD);

  //printf("range h = %10.2e ~ %10.2e, in rank %d %d %d\n",
  //hmin, hmax, thisid[0], thisid[1], thi    sid[2]);

  float dtmax = 1.3 * hmin_global / vp_max_global;

  if(masternode){
    printf("global range of h   = %10.2e ~ %10.2e m\n",
        hmin_global, hmax_global);
    printf("global range of vp  = %10.2e ~ %10.2e m/s\n",
        vp_min_global, vp_max_global);
    printf("global range of vs  = %10.2e ~ %10.2e m/s\n",
        vs_min_global, vs_max_global);
    printf("global range of rho = %10.2e ~ %10.2e kg/m^3\n",
        rho_min_global, rho_max_global);
    if(hostParams.DT < dtmax){
      printf("DT = %10.2e < dtmax = %10.2e (sec)\n"
             "satisfy stability condition, OK\n",
          hostParams.DT, dtmax);
    }else{
      printf("Serious Error: DT = %10.2e > dtmax = %10.2e (sec)\n"
             "               do not satisfy stability condition, ABORT!\n",
          hostParams.DT, dtmax);
      MPI_Barrier(MPI_COMM_WORLD);
      MPI_Abort(MPI_COMM_WORLD, 110);
    }
  }

  CUDACHECK(hipDeviceSynchronize());
  CUDACHECK(hipGetLastError());
  
  hipMalloc((int**) &Faultgrid, sizeof(int)*4*hostParams.num_fault);
  hipMemcpy(Faultgrid, hostParams.Fault_grid, sizeof(int)*4*hostParams.num_fault, hipMemcpyHostToDevice);
  // for (int i = 0; i < 4*hostParams.num_fault; i++){
  //   Faultgrid[i] = hostParams.Fault_grid[i];
  // }
#ifdef Rupture
  for(int nfault = 0; nfault < num_fault; nfault++){
    i0 = hostParams.src_i[nfault];
    init_fault_coef(M, F, i0, nfault);
    init_fault(F, nfault, Faultgrid);
  }
  if (1 == hostParams.INPORT_STRESS_TYPE){
    // overwrite by input init stress
    nc_read_init_stress(F);
  }
#endif
#ifdef FreeSurface
  init_wave_free(M, P, W);
#endif

  CUDACHECK(hipDeviceSynchronize());
  CUDACHECK(hipGetLastError());

  /* for timing */
  double t_start;
  double t_elapsed;
  double elapsed_time_s;
  elapsed_time_s = 0.f;

  hipEvent_t e_start, e_end;

  hipEventCreate(&e_start);
  hipEventCreate(&e_end  );

#ifdef useNetCDF
#ifdef Rupture
  ncFile ncFault;
#endif
  ncFile ncSliceX;
  ncFile ncSliceY;
  ncFile ncSliceZ;
 // ncFile ncRecv;
#endif

  int nt = hostParams.NT;
  init_wave_3d(W.W);
#ifdef usePML
  int ND = hostParams.PML_N;
  if(P.isx1) hipMemset(P.Wx1, 0, sizeof(real_t)*ND*nj*nk);
  if(P.isx2) hipMemset(P.Wx2, 0, sizeof(real_t)*ND*nj*nk);
  if(P.isy1) hipMemset(P.Wy1, 0, sizeof(real_t)*ND*nk*ni);
  if(P.isy2) hipMemset(P.Wy2, 0, sizeof(real_t)*ND*nk*ni);
  if(P.isz1) hipMemset(P.Wz1, 0, sizeof(real_t)*ND*ni*nj);
  if(P.isz2) hipMemset(P.Wz2, 0, sizeof(real_t)*ND*ni*nj);
#endif
  hipDeviceSynchronize();
  CUDACHECK(hipGetLastError());

  int slice_x_index = hostParams.EXPORT_WAVE_SLICE_X;
  int slice_y_index = hostParams.EXPORT_WAVE_SLICE_Y;
  int slice_z_index = hostParams.EXPORT_WAVE_SLICE_Z;
#ifdef useNetCDF
#ifdef Rupture
//  printf("aaaaaa\n");
  nc_def_fault(F, &ncFault);
//  printf("Netcdf def OK\n");
  nc_put_fault_coord(hostCoord, ncFault, faultnode);
  //CUDACHECK(hipDeviceSynchronize()); MPI_Barrier(MPI_COMM_WORLD); MPI_Finalize(); return 0;

  int thread_count = 2;
  pthread_t* thread_handles = (pthread_t*)malloc(thread_count * sizeof(pthread_t));

  size_t ibytes = sizeof(real_t)*nj*nk*num_fault;
  real_t *hostFault = NULL;
  hipHostMalloc((void**)&hostFault, 16*ibytes);
  // hostFault = (real_t*)malloc(16*ibytes);
  real_t *deviceFault = NULL;
  hipMalloc((void **)&deviceFault, 16*ibytes);

  real_t *hostData = NULL;
  real_t *deviceData = NULL;
  int countwave = 3*sizeof(real_t)*(ni*nj+ni*nk+nk*nj);
  hipMalloc((void **) &deviceData, countwave);
  hipHostMalloc((void**)&hostData, countwave);
  real_t* hostptr[4] = {hostFault, hostData, deviceFault, deviceData};

#endif
  nc_def_wave_xy(slice_z_index, &ncSliceZ);
  nc_put_wave_xy_coord(hostCoord, slice_z_index, ncSliceZ);

  nc_def_wave_xz(slice_y_index, &ncSliceY);
  nc_put_wave_xz_coord(hostCoord, slice_y_index, ncSliceY);

  nc_def_wave_yz(slice_x_index, &ncSliceX);
  nc_put_wave_yz_coord(hostCoord, slice_x_index, ncSliceX);

//  nc_def_recv(R, W, &ncRecv);
#endif
  for (int it = 0; it < nt; it++){

    /* for timing */
    t_start = seconds();
    /* for timing */
#ifdef useNetCDF
    int time_skip = hostParams.EXPORT_TIME_SKIP;
    int it_skip;
    if ((it % time_skip) == 0){
      it_skip = (int)(it/time_skip);
#ifdef Rupture
    if (it_skip > 0)
      {
        pthread_join(thread_handles[0], NULL);
        pthread_join(thread_handles[1], NULL);
      }

    nc_put_faultwave(F, W.W, slice_z_index, slice_y_index, slice_x_index, hostptr);
    pthreadFault pthdFault = {hostptr[0], it_skip, hostParams.nj, hostParams.nk, ncFault};
    pthreadWave pthdWave = {hostptr[1], it_skip, hostParams.ni, hostParams.nj, hostParams.nk,\
                            ncSliceX, ncSliceY, ncSliceZ, slice_x_index, slice_y_index, slice_z_index,\
                            {thisid[0], thisid[1], thisid[2]}};

    pthread_create(&thread_handles[0], NULL, ncput_fault, &pthdFault);
    pthread_create(&thread_handles[1], NULL, ncput_wave, &pthdWave);
//      printf("put OK \n");
#else
// #endif
    nc_put_wave_xy(W.W, slice_z_index, it_skip, ncSliceZ);
    nc_put_wave_xz(W.W, slice_y_index, it_skip, ncSliceY);
    nc_put_wave_yz(W.W, slice_x_index, it_skip, ncSliceX);
#endif
    }
#ifdef Rupture
    if ( (it % time_skip) == 0 && (it == (nt-1) || it == (nt-2)) )
    {
      pthread_join(thread_handles[0], NULL);
      pthread_join(thread_handles[1], NULL);
    }
#endif
#endif

    // select flags for F(orward) or B(ackward) operator
    int FlagX = FD_Flags[it % 8][0];
    int FlagY = FD_Flags[it % 8][1];
    int FlagZ = FD_Flags[it % 8][2];

    for(int nfault = 0; nfault < num_fault; nfault++){
      cal_rup_sensor(F, nfault, Faultgrid);
      if (hostParams.Friction_type == 3){
        thermpress(W, F, M, it, 0, FlagX, FlagY, FlagZ, nfault);
        }
    }
    for (int irk = 0; irk < 4; irk ++){

      //nc_put_fault(F, it*4+irk, ncFault);
      //nc_put_fault(F, it, ncFault);
      //if(irk % 2){ FlagX *= -1; FlagY *= -1; FlagZ *= -1;}

      exchange_wave(W.W);
#ifdef Rupture
    for(int nfault = 0; nfault < num_fault; nfault++){
      i0 = hostParams.src_i[nfault];
      exchange_fault(F, nfault);
      wave2fault(W, F, M, i0, nfault);
      state_rk(F, irk, nfault, Faultgrid);
      if (hostParams.Friction_type == 0){
        trial_sw(W, F, M, it, irk, FlagX, FlagY, FlagZ, i0, nfault, Faultgrid);
      }else if (
          hostParams.Friction_type == 1 ||
          hostParams.Friction_type == 2 ||
          hostParams.Friction_type == 3 ){
        trial_rs(W, F, M, it, irk, FlagX, FlagY, FlagZ, i0, nfault, Faultgrid);
      }
      
      fault2wave(W, F, M, i0, nfault);
    }
#endif
      wave_deriv(W, M, P, FlagX, FlagY, FlagZ);
#ifndef Rupture
      add_source_ricker(W, M, it, irk);
#endif
#ifdef DynTrigger
      add_source_ricker(W, M, it, irk);
#endif
#ifdef usePML
      if(h_P.isx1) abs_deriv_x(W, M, P, FlagX, 0);
      if(h_P.isx2) abs_deriv_x(W, M, P, FlagX, ni-ND);

      if(h_P.isy1) abs_deriv_y(W, M, P, FlagY, 0);
      if(h_P.isy2) abs_deriv_y(W, M, P, FlagY, nj-ND);

      if(h_P.isz1) abs_deriv_z(W, M, P, FlagZ, 0);
      if(h_P.isz2) abs_deriv_z(W, M, P, FlagZ, nk-ND);
#endif

#ifdef Rupture
      for(int nfault = 0; nfault < num_fault; nfault++){
        i0 = hostParams.src_i[nfault];
        //fault_deriv(W, F, M, FlagX, FlagY, FlagZ);
        fault_dvelo(W, F, M, FlagX, FlagY, FlagZ, i0, nfault);
        //fault_dstrs(W, F, M, FlagX, FlagY, FlagZ);
        if(FlagX == 1){ // Forward
          fault_dstrs_f(W, F, M, FlagX, FlagY, FlagZ, i0, nfault);
        }else{
          fault_dstrs_b(W, F, M, FlagX, FlagY, FlagZ, i0, nfault);
        }
        // smoothT1(F, nfault, Faultgrid);
      }

#endif
      wave_rk(W, P, irk);

#ifdef Rupture
  for(int nfault = 0; nfault < num_fault; nfault++){
      i0 = hostParams.src_i[nfault];
      fault_rk(F, irk, nfault);
//#if defined(Rupture) && defined(FaultSmooth)
//      exchange_wave(W.W);
//      exchange_fault(F);
//      smooth_gauss_volume(W, F, M);
//#endif
      fault2wave(W, F, M, i0, nfault);
  }
#endif

#ifdef useCerjan
      apply_cerjan(W, damp);
#endif

      hipDeviceSynchronize();
      //CUDACHECK(hipGetLastError());

      // Reverse the FD Flags for next substep of Runge-Kutta
      FlagX *= -1; FlagY *= -1; FlagZ *= -1;
    } // end irk
#if defined(Rupture) && defined(FaultSmooth)
    if (it % 1 == 0){
    exchange_wave(W.W);
    for(int nfault = 0; nfault < num_fault; nfault++){
      i0 = hostParams.src_i[nfault];
      exchange_fault(F, nfault);
      smooth_gauss_volume(W, F, M, i0, nfault, Faultgrid, it);
    }
    //fault_filter(W, F, M);
    }
#endif
    // if (it % 1 == 0){
    //   exchange_wave(W.W);
    //   for(int nfault = 0; nfault < num_fault; nfault++){
    //     i0 = hostParams.src_i[nfault];
    //     exchange_fault(F, nfault);
    //     smooth_T1(F, nfault, Faultgrid);
    // }
    // }
   
    /* for timing */
    hipDeviceSynchronize();
    t_elapsed = seconds() - t_start;
    elapsed_time_s += t_elapsed;
    if(0 == it % 10 && masternode){
      printf("> %6d, total %d, %8.4lf s\n", it, nt, t_elapsed);
      fflush(stdout);
    }

  } // end it

#ifdef useNetCDF
#ifdef Rupture
  nc_end_fault(ncFault);
#endif
  nc_end_wave_xz(slice_y_index, ncSliceY);
  nc_end_wave_xy(slice_z_index, ncSliceZ);
  nc_end_wave_yz(slice_x_index, ncSliceX);

 // nc_end_recv(R, ncRecv);
#endif

  /* for timing */
  float elapsed_time_ms_cuda;
  hipEventRecord(e_end, 0);
  hipEventElapsedTime(&elapsed_time_ms_cuda, e_start, e_end);
  hipDeviceSynchronize();
  hipGetLastError();
  fflush(stdout);
  if(masternode){
    printf("----------------------------------\n");
    printf("> Total %6d: CPU: %e s | GPU: %e s\n",
        nt, elapsed_time_s, elapsed_time_ms_cuda/1e3);
  }
  //printf("> Performance: %8.2f MCells/s\n",
  //    (double) nx * ny * nz / (elapsed_time_ms * 1e3f));
  fflush(stdout);

  hipEventDestroy(e_start);
  hipEventDestroy(e_end);
  /* for timing */

  CUDACHECK(hipDeviceSynchronize());
  CUDACHECK(hipGetLastError());

  dealloc_wave(W);
  //dealloc_fault(F);
  //dealloc_fault_coef(F);
  ////dealloc_pml_host(h_P);
  //dealloc_pml(P);
  hipFree(Faultgrid);
  hipFree(deviceData);
  hipFree(deviceFault);
  hipHostFree(hostFault);
  hipHostFree(hostData);

  MPI_Finalize();

  return 0;
}
